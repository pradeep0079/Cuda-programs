
#include<stdio.h>
#include <hip/hip_runtime.h>
#include <stdlib.h>

#define BS 8
#define N 10

void print(int *A, int n){
    for(int i = 0; i < n; i++){
        printf("%d ", A[i]);
    }
    printf("\\n");
}

__global__ void increment(int *A, int n){
    int i = blockDim.x * blockIdx.x + threadIdx.x;
    if(i < n) A[i]++;
}

int main (){
    int threadsPerBlock, blocksPerGrid, n, *A, *dA;
    n = N;
    threadsPerBlock = BS;
    blocksPerGrid = (n + BS - 1) / BS;
    A = (int *)malloc(n * sizeof(int));
    for (int i = 0; i < n; i++){
        A[i] = i * 10; // 0, 10, 20, 30, 40, 50, 60, 70, 80, 90
    }

    hipMalloc((void **)&dA, n * sizeof(int));
    hipMemcpy(dA, A, n * sizeof(int), hipMemcpyHostToDevice);
    increment<<<blocksPerGrid, threadsPerBlock>>>(dA, n);

    hipMemcpy(A, dA, n * sizeof(int), hipMemcpyDeviceToHost);
    print(A, n);
    hipFree(dA);
    free(A);
    return 0;
}





